/**
 *
 *  This is a cuda version of the array addition program as created from the 
 *  tutorial from here:
 *
 *  https://devblogs.nvidia.com/even-easier-introduction-cuda/
 *
 *  Any adjustments made are made from suggestions from Programming Massively
 *  Parallel Processors, 3rd Edition:
 *
 *  https://www.amazon.com/Programming-Massively-Parallel-Processors-Hands/dp/0128119861/ref=dp_ob_title_bk
 *
 * */

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

            // Signifies a kernel function
__global__  // Runs on device code
void deviceAdd(int n, float *dst, float *src)
{
  int index = blockDim.x * blockIdx.x + threadIdx.x;

  if(index < n) {
    dst[index] = src[index] + dst[index];
  }
}

void add(int n, float* h_dst, float* h_src) {
  int size = n * sizeof(float);
  float *d_dst, *d_src;
  
  // This allocates memory and copies 
  // the memory from the host to the 
  // device memory.
  hipMalloc((void **) &d_src, size);
  hipMemcpy(d_src, h_src, size, hipMemcpyHostToDevice);
  hipMalloc((void **) &d_dst, size);
  hipMemcpy(d_dst, h_dst, size, hipMemcpyHostToDevice);

  deviceAdd<<<ceil(n / 256.0), 256>>>(n, d_dst, d_src);

  hipMemcpy(h_dst, d_dst, size, hipMemcpyDeviceToHost);
  
  hipFree(d_src);
  hipFree(d_dst);
}

int main(void)
{
  int N = 100<<20; // 100M elements

  float* x = (float*) malloc(N * sizeof(float));
  float* y = (float*) malloc(N * sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the CPU
  add(N, x, y);

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  free(x);
  free(y);

  return 0;
}
